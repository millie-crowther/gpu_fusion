#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

/*
   Device code
 */

__device__ float voxel_length = 20.0f;
__device__ float eta = 0.1f;
__device__ float delta = 200.0f;

__device__ int
get_global_id(){
    return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ float
distance(float * phi, float x, float y, float z, int width, int height){
    if (
	x < 0 || y < 0 ||
	x / voxel_length <= width ||
	y / voxel_length <= height
    ){
	return 1.0f;
    }
    int index = x / voxel_length + y / voxel_length * width;
    float phi_true = phi[index] - z;

    // divide by delta
    float result = phi_true / delta;
    
    // clamp to range [-1..1]
    if (result > 1.0f){
	return 1.0f;
    } else if (result < -1.0f){
	return -1.0f;
    } else {
	return result;
    }
}

__global__ void
init_kernel(
    float * phi, 
    float * phi_global, 
    int width, int height, int depth
){
    // calculate unique id
    int id = get_global_id();
    if (id >= width * height * depth){
	return;
    }

    // get coordinates from id
    int x  = (id % (width * height)) % width;
    int y  = (id % (width * height)) / width;
    int z  = id / (width * height);

    // sample sdf
    phi_global[id] = phi[x + y * width] - z * voxel_length;
}

__global__ void
rigid_kernel(
    float * phi, 
    float * phi_global, 
    float * u, float * v, float * w, 
    int width, int height, int depth
){
    // calculate unique id
    int id = get_global_id();
    if (id >= width * height * depth){
	return;
    }

    // get coordinates from id
    int x  = (id % (width * height)) % width;
    int y  = (id % (width * height)) / width;
    int z  = id / (width * height);

    float px = x * voxel_length;
    float py = y * voxel_length;
    float pz = z * voxel_length;
   
    // rigid update
    float s = phi[x + y * height] - phi_global[id];
    float dx = (
        distance(phi, px + voxel_length, py, pz, width) - 
	distance(phi, px - voxel_length, py, pz, width)
    ) / (2.0f * voxel_length); 

    float dy = (
        distance(phi, px, py + voxel_length, pz, width) - 
	distance(phi, px, py - voxel_length, pz, width)
    ) / (2.0f * voxel_length); 

    float dz = (
        distance(phi, px, py, pz + voxel_length, width) - 
	distance(phi, px, py, pz - voxel_length, width)
    ) / (2.0f * voxel_length); 

    // perform update
    u[id] -= eta * s * dx;
    v[id] -= eta * s * dy;
    w[id] -= eta * s * dz;
}

__global__ void
nonrigid_kernel(
    float * phi, 
    float * phi_global, 
    float * u, float * v, float * w, 
    int width, int height, int depth
){
    // calculate unique id
    int id = get_global_id();
    if (id >= width * height * depth){
	return;
    }

    // get coordinates from id
    int x  = (id % (width * height)) % width;
    int y  = (id % (width * height)) / width;
    int z  = id / (width * height);

    // rigid update
    float s = phi[x + y * height] - phi_global[id];
    float dx = (
        distance(phi, px + voxel_length, py, pz, width) - 
	distance(phi, px - voxel_length, py, pz, width)
    ) / (2.0f * voxel_length); 

    float dy = (
        distance(phi, px, py + voxel_length, pz, width) - 
	distance(phi, px, py - voxel_length, pz, width)
    ) / (2.0f * voxel_length); 

    float dz = (
        distance(phi, px, py, pz + voxel_length, width) - 
	distance(phi, px, py, pz - voxel_length, width)
    ) / (2.0f * voxel_length); 

    // perform update
    u[id] -= eta * s * dx;
    v[id] -= eta * s * dy;
    w[id] -= eta * s * dz;

    //TODO: level set energy and killing energy
}


/*
   Host code
 */
int grid_size = 512;
int block_size = 512;

void
initialise(
    float * phi,
    float ** device_phi,  
    float ** phi_global,
    float ** u, float ** v, float ** w,
    int width, int height, int depth,
){
    int vol_size = sizeof(float) * width * height * depth;
    int img_size = sizeof(float) * width * height;

    // allocate memory
    hipMalloc(phi_global, vol_size);
    hipMalloc(u, vol_size);
    hipMalloc(v, vol_size);
    hipMalloc(w, vol_size);

    // sample sdf into canon sdf
    hipMalloc(device_phi, img_size);
    hipMemcpy(*device_phi, phi, img_size, hipMemcpyHostToDevice);
    init_kernel<<<grid_size, block_size>>>(
        *device_phi, *phi_global, width, height, depth
    );

    // set deform field to zero
    hipMemset(*u, 0, vol_size);
    hipMemset(*v, 0, vol_size);
    hipMemset(*w, 0, vol_size);
}

void 
update_rigid(
    float * phi,
    float * device_phi, 
    float * phi_global, 
    float * u, float * v, float * w, 
    int width, int height, int depth,
){
    hipMemcpy(device_phi, phi, sizeof(float) * width * height, hipMemcpyHostToDevice);

    rigid_kernel<<<grid_size, block_size>>>(
        device_phi, phi_global, u, v, w, width, height, depth
    );
}

void 
update_nonrigid(
    float * phi, 
    float * device_phi, 
    float * phi_global, 
    float * u, float * v, float * w,
    int width, int height, int depth,
){
    hipMemcpy(device_phi, phi, sizeof(float) * width * height, hipMemcpyHostToDevice);

    nonrigid_kernel<<<grid_size, block_size>>>(
	device_phi, phi_global, u, v, w, width, height, depth
    );
}

void
get_canon(float * host_phi_global, float * device_phi_global, int size){
    hipMemcpy(host_phi_global, device_phi_global, size, hipMemcpyDeviceToHost);
}
         
void cleanup(float * phi_global, float * u, float * v, float * w, float * device_phi){
    hipFree(phi_global);
    hipFree(u);
    hipFree(v);
    hipFree(w);
    hipFree(device_phi);
}

